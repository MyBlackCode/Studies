
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main() {
	
	int n, * a, * b, *c;
	int *a_d, *b_d, *c_d;
	
	std::cout << "Podaj n:";
	std:: cin >> n;
	
	a = new int [n];
	b = new int [n];
	c = new int [n];
	
	for(int i = 0; i< n; i++)
	{
		a[i] = i;
		b[i] = 10;
		c[i] = 0;
	}
	hipMalloc((void **)&a_d, n * sizeof(int));
	hipMalloc((void **)&b_d, n * sizeof(int));
	hipMalloc((void **)&c_d, n * sizeof(int));
	
	hipMemcpy(a_d, a, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(c_d, c, n*sizeof(int), hipMemcpyHostToDevice);	
	
	add<<<1,n>>>(a_d, b_d, c_d);
	//for(int i = 0; i< n; i++)
	//{
	//	c[i] = a[i] + b[i];
	//}
	
	hipMemcpy(c, c_d, n*sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i< n; i++)
	{
		std::cout << a[i]  <<" + " << b[i] << " = " << c[i] << std::endl;
	}
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
	
	delete[] a;
	delete[] b;
	delete[] c;
	return 0;
}
